#include <hip/hip_runtime.h>

__global__ void diag_matmul_kernel(
    const float* diag,
    const float* mat,
    float* out,
    const int N,
    const int M) {
    
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < M) {
        out[row * M + col] = diag[row] * mat[row * M + col];
    }
}
