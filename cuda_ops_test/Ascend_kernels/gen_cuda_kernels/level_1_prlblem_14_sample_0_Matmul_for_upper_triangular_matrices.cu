#include <hip/hip_runtime.h>
__global__ void triu_matmul_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N && row <= col) {
        float sum = 0.0f;
        for (int k = row; k <= col; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
