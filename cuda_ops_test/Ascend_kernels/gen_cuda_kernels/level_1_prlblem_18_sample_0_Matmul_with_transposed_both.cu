#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define BLOCK_SIZE 16

__global__ void MatMulKernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int M, int N, int K) {

