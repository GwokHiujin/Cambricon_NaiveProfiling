#include <hip/hip_runtime.h>
__global__ void swish_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float sigmoid = 1.0f / (1.0f + expf(-x));
        output[idx] = x * sigmoid;
    }
}
