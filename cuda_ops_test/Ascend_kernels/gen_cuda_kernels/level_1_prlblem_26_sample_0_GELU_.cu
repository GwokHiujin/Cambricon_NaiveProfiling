#include <hip/hip_runtime.h>
#include <cmath>
__global__ void gelu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float cdf = 0.5f * (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
        output[idx] = x * cdf;
    }
}
