#include <hip/hip_runtime.h>

__global__ void hardtanh_kernel(const float* input, float* output, int size, float min_val, float max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = input[idx];
        output[idx] = val < min_val ? min_val : (val > max_val ? max_val : val);
    }
}
