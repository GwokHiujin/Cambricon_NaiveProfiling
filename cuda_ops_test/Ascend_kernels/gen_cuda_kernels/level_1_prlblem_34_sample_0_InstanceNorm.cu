#include <hip/hip_runtime.h>

__global__ void compute_mean_var_kernel(
    const float* __restrict__ x,
    float* mean,
    float* var,
    int N, int C, int H, int W) {
    int n = blockIdx.x;
    int c = blockIdx.y;
    int HW = H * W;

    extern __shared__ float shared_data[];
    float* s_sum = shared_data;
    float* s_sum2 = shared_data + blockDim.x;

    float sum = 0.0f;
    float sum2 = 0.0f;

    int thread_idx = threadIdx.x;
    int num_threads = blockDim.x;

    for (int i = thread_idx; i < HW; i += num_threads) {
        int index = ((n * C + c) * H * W) + i;
        float val = x[index];
        sum += val;
        sum2 += val * val;
    }

    s_sum[thread_idx] = sum;
    s_sum2[thread_idx] = sum2;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_idx < s) {
            s_sum[thread_idx] += s_sum[thread_idx + s];
            s_sum2[thread_idx] += s_sum2[thread_idx + s];
        }
        __syncthreads();
    }

    if (thread_idx == 0) {
        float mean_val = s_sum[0] / HW;
        float var_val = s_sum2[0] / HW - mean_val * mean_val;
        mean[n * C + c] = mean_val;
        var[n * C + c] = var_val;
    }
}
