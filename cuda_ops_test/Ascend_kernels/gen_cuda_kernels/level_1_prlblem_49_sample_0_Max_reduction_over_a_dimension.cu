#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void max_reduction_kernel(const float* x, float* out, int dim_size, int stride, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        int batch_idx = idx / stride;
        int element_idx = idx % stride;
        
        float max_val = -FLT_MAX;
        for (int i = 0; i < dim_size; ++i) {
            float val = x[batch_idx * dim_size * stride + i * stride + element_idx];
            if (val > max_val) {
                max_val = val;
            }
        }
        out[idx] = max_val;
    }
}
