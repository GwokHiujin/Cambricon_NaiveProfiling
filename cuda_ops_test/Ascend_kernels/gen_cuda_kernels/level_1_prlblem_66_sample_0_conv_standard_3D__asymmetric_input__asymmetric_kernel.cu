#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void conv3d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int depth_in,
    int height_in,
    int width_in,
    int depth_out,
    int height_out,
    int width_out,
    int kernel_d,
    int kernel_h,
    int kernel_w,
    int stride_d,
    int stride_h,
    int stride_w,
    int padding_d,
    int padding_h,
    int padding_w,
    int dilation_d,
    int dilation_h,
    int dilation_w,
    int groups
) {

