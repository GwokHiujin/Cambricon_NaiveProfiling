#include <hip/hip_runtime.h>

// Tile size for shared memory
#define TILE_SIZE 16

__global__ void tall_skinny_matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B, 
    float* __restrict__ C,
    const int M, const int N) {

    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int i = 0; i < N; i += TILE_SIZE) {
        if (row < M && (i + tx) < N)
            As[ty][tx] = A[row * N + (i + tx)];
        else
            As[ty][tx] = 0.0f;
            
        if ((i + ty) < N && col < M)
            Bs[ty][tx] = B[(i + ty) * M + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    
    if (row < M && col < M) {
        C[row * M + col] = sum;
    }
}
